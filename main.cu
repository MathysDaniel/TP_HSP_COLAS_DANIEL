#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

void MatrixInit(float *M, int n, int p){
    for (int i = 0; i < n; i++){
        for (int j = 0; j < p; j++){
            *(M + i * p + j) = (float)(rand() % 200) / 100 - 1;
        }
    }
}

void MatrixPrint(float *M, int n, int p){
    for(int x = 0 ; x < n ; x++) {
        printf(" (");
        for(int y = 0 ; y < p ; y++){
            printf("%f     ", *(M + x * p + y));
        }
        printf(")\n");
    }
}

void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < p; j++) {
            *(Mout + i * p + j) = *(M1 + i * p + j) + *(M2 + i * p + j);
        }
    }
}

__global__ void cudaMatrixAdd(float *M1, float *M2, float *Mout, int n, int p) {
    int row = blockIdx.x;    // Fetch the block index as row
    int col = threadIdx.x;   // Fetch the thread index as column

    if (row < n && col < p) {
        int index = row * p + col;  // Calculate the index in the flattened array

        // Perform addition
        Mout[index] = M1[index] + M2[index];
    }
}

void MatrixMult(float *M1, float *M2, float *Mout, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            float sum = 0.0;
            for (int k = 0; k < n; k++) {
                sum += M1[i * n + k] * M2[k * n + j];
            }
            Mout[i * n + j] = sum;
        }
    }
}

__global__ void cudaMatrixMult(float *M1, float *M2, float *Mout, int n) {
    int row = blockIdx.x;    
    int col = threadIdx.x;

    if (row < n && col < n) {
        float sum = 0.0;
        for (int i = 0; i < n; i++) {
            sum += M1[row * n + i] * M2[i * n + col];
        }
        Mout[row * n + col] = sum;
    }
}

int main(void){
    int n = 5;
    int p = 5;
    float *M = (float *)malloc(n * p * sizeof(float));

    float *CPU_M1 = (float *)malloc(n * p * sizeof(float));
    float *CPU_M2 = (float *)malloc(n * p * sizeof(float));
    float *CPU_Mout = (float *)malloc(n * p * sizeof(float));

    MatrixInit(M, n, p);

    MatrixPrint(M, n, p);

    clock_t start_cpu, end_cpu;
    double cpu_time_used;

    // Addition with CPU
    start_cpu = clock();
    MatrixAdd(CPU_M1, CPU_M2, CPU_Mout, n, p);
    end_cpu = clock();
    cpu_time_used = ((double)(end_cpu - start_cpu)) / CLOCKS_PER_SEC;

    printf("Time taken for CPU addition: %f seconds\n", cpu_time_used);

    free(CPU_M1);
    free(CPU_M2);
    free(CPU_Mout);

    // Addition with GPU
    float *GPU_M1, *GPU_M2, *GPU_Mout;
    hipMalloc((void **)&GPU_M1, n * n * sizeof(float));
    hipMalloc((void **)&GPU_M2, n * n * sizeof(float));
    hipMalloc((void **)&GPU_Mout, n * n * sizeof(float));



    dim3 gridDim(n, 1, 1);   // Each block handles one row
    dim3 blockDim(p, 1, 1);  // Each block has threads equal to columns

    cudaMatrixAdd<<<gridDim, blockDim>>>(GPU_M1, GPU_M2, GPU_Mout, n, p);

    hipFree(GPU_M1);
    hipFree(GPU_M2);
    hipFree(GPU_Mout);

    float *CPU_M3 = (float *)malloc(n * p * sizeof(float));
    float *CPU_M2 = (float *)malloc(n * p * sizeof(float));
    float *CPU_Mout2 = (float *)malloc(n * p * sizeof(float));

    // Multiplication with CPU
    MatrixMult(ptr_M1, ptr_M2, ptr_Mout, n);

    free(CPU_M3)
    free(CPU_M4)
    free(CPU_Mout2)

    // Multiplication with GPU
    float *GPU_M3, *GPU_M4, *GPU_Mout2;
    hipMalloc((void **)&GPU_M3, n * n * sizeof(float));
    hipMalloc((void **)&GPU_M4, n * n * sizeof(float));
    hipMalloc((void **)&GPU_Mout2, n * n * sizeof(float));

    dim3 gridDim(n, 1, 1);   
    dim3 blockDim(n, 1, 1);  

    cudaMatrixMult<<<gridDim, blockDim>>>(GPU_M3, GPU_M4, GPU_Mout2, n);

    hipFree(GPU_M3);
    hipFree(GPU_M4);
    hipFree(GPU_Mout2);

    free(M); 
    return 0;
}